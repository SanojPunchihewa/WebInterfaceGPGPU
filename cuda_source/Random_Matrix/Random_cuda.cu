/*Author : H.M.Gamaarachchi
Generate random floating point matrix using CUDA */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "myhelpers.h"

int main(int argc,char *argv[]){

	//check argc
	if (argc<5){
		fprintf(stderr,"Please enter all arguments eg : ./random rows cols seed file.txt\n");
		exit(EXIT_FAILURE);
	}
	
	//var declaration
	int width=atoi(argv[2]);
	int height=atoi(argv[1]);
	int seed=atoi(argv[3]);
	hiprandGenerator_t generator;
	float *data, *dev_data;
	
	//mem allocation
	data=(float *)malloc(sizeof(float)*width*height);
	isMemoryFull(data);
	checkCudaError(hipMalloc((void **)&dev_data,sizeof(float)*width*height));
	
	
	//generate random numbers
	hiprandCreateGenerator(&generator,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(generator,seed);
	checkCudaError(hipGetLastError());
	
	//Time
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);	
	
	hiprandGenerateUniform(generator,dev_data,width*height);
	checkCudaError(hipGetLastError());
	
	//time
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Time spent for calculation in CUDA : %.10f s\n",elapsedtime/(float)1000);	
	
	//memcopy
	checkCudaError(hipMemcpy(data,dev_data,sizeof(float)*width*height,hipMemcpyDeviceToHost));
	
	//writing to file
	FILE *fp=fopen(argv[4],"w");
	isFileOK(fp);
	int i,j;
	for (i=0;i<height;i++){
		for (j=0;j<width;j++){
			fprintf(fp,"%f ",data[i*width+j]*10);
		}
		fprintf(fp,"\n");
	}	
	
	//free
	hipFree(dev_data);
	hiprandDestroyGenerator(generator);
	free(data);

	return 0;
}
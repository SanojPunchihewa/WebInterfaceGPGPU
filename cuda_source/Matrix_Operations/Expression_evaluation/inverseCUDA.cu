/*author : H.M.Gamaarachchi
c file for solving linear equations using LU decomposition
reference : http://rosettacode.org/wiki/LU_decomposition#C
http://www.ece.mcmaster.ca/~kiruba/3sk3/lecture6.pdf
 A.x[i th col]=b[ith col] solved by forward substitution L.d[ith col]=b[ith col] and the backward substitution U.x[ith col]=d[ith col]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <stdlib.h>
#include <time.h>

/*
//print an array
void printarray(float *mat,int width,int height){
	int i,j;
	for (i=0;i<height;i++){
		for (j=0;j<width;j++){
			printf("%f ",mat[width*i+j]);
			}
		printf("\n");
	}
}*/

//finding highest factor <=maxsize
int hf(int size,int maxsize){ 
  int i=1;int factor=1;
  for (i=2;i<=maxsize;i++){
	if(size%i==0){
		factor=i;
	}
  }
  return factor;
}


//initialize matrixes
__global__ void initialize(float *matL,float *matU,float *matb, int size){
	int i,j;
	
	i= blockIdx.y * blockDim.y + threadIdx.y;
	j= blockIdx.x * blockDim.x + threadIdx.x;

	//make matL to an identity matrix and matU to a zero matrix
	matU[i*size+j]=0;
	matL[i*size+j]=0;
	matb[i*size+j]=0;
	if (i==j){
		matL[i*size+j]=1;
		matb[i*size+j]=1;
	}


}

	
//do LU decomposition in parrel nth row nth col
__global__ void LUdecompose_CUDA(float *matL,float *matU,float *matA,int size,int current){

	int i,j,k,n;
	float u,l;
	n= blockIdx.x * blockDim.x + threadIdx.x;
	
	if(n>=current && n<size){
	
		//calculating uppers in ith row
		i=current; 
		j=n;
		u=matA[i*size+j]; //a(i,j)
		l=matA[i*size+j];	
		if (j>=i){
			for(k=0;k<i;k++){
				u=u-matU[k*size+j]*matL[i*size+k]; //u(i,j)=a(i,j)-sigma(u(k,j)*l(i,k))
				}
			matU[i*size+j]=u; //substitute calculated uppers
		}

		//calculating lowers ith colum
		i=n;
		j=current;
		u=matA[i*size+j]; //a(i,j)
		l=matA[i*size+j];	
		if (j<=i){
			for(k=0;k<j;k++){
				l=l-matU[k*size+j]*matL[i*size+k];	
			}
			matL[i*size+j]=l/(float)matU[j*size+j];  //substitute calculated lowers
		}
	}
			
}	

//linear solving to get inverse
__global__ void linearSolve(float *inverse, float *matL, float *matU,float *matb,float *tempd, int size){
	int i,j,col_mat;
	float d,x;
	
	col_mat = blockIdx.x * blockDim.x + threadIdx.x;

		//forward substitution
		for (i=0;i<size;i++){
			d=matb[i*size+col_mat]; //initially di=bi
			for (j=0;j<i;j++){
				d=d-matL[i*size+j]*tempd[j*size+col_mat]; //di=bi-sigma(Lij*dj)
			}
			tempd[i*size+col_mat]=d;
		}
		
		//backward substitution
		for (i=size-1;i>=0;i--){
			x=tempd[i*size+col_mat]; //initially xi=di
			for (j=i+1;j<size;j++){
				x=x-matU[i*size+j]*inverse[j*size+col_mat]; //xi=di-sigma(Uij*xj)
			}
			inverse[i*size+col_mat]=x/matU[i*size+i]; //xi=xi/Uii
		}	

}

//Inversion abstraction	
void inverse(float *inverse,float *matA,int size){
	float *dev_L,*dev_U,*dev_A,*dev_b,*dev_d,*dev_inverse;
	
	//allocating memory on device
	hipMalloc((void**)&dev_A, size*size*sizeof(float)); //original matrix to find inverse
	hipMalloc((void**)&dev_L, size*size*sizeof(float));
	hipMalloc((void**)&dev_U, size*size*sizeof(float));
	hipMalloc((void**)&dev_b, size*size*sizeof(float));	
	hipMalloc((void**)&dev_d, size*size*sizeof(float));
	hipMalloc((void**)&dev_inverse, size*size*sizeof(float));

	//Copying data to device
	hipMemcpy(dev_A,matA, size*size*sizeof(float),hipMemcpyHostToDevice);
	
	//thread distribution for initialization
	int highest_size=hf(size,32);
	dim3 grid(ceil(size/highest_size),ceil(size/highest_size));
	dim3 block(highest_size,highest_size);

	//thread distribution for calculation
	int block1=hf(size,32);
	int grid1=ceil(size/block1);

	//Time
	//cudaEvent_t start,stop;
	//float elapsedtime;
	//cudaEventCreate(&start);
	//cudaEventRecord(start,0);	

	//function
	initialize<<<grid,block>>>(dev_L,dev_U,dev_b, size); //initialization
	hipDeviceSynchronize();

	int i=0;
	for (i=0;i<size;i++){
		//decomposition
		LUdecompose_CUDA<<<grid1,block1>>>(dev_L,dev_U,dev_A,size,i);
		hipDeviceSynchronize();
	}
	
	linearSolve<<<grid1,block1>>>(dev_inverse, dev_L, dev_U, dev_b, dev_d,size); //solving linear equations
	hipDeviceSynchronize();

	//Time
	/*cudaEventCreate(&stop);
	cudaEventRecord(stop,0);
	cudaEventSynchronize(stop);
	cudaEventElapsedTime(&elapsedtime,start,stop);
	printf("Time spent for calculation : %.10f\n",elapsedtime/(float)1000);*/
	
	//copyback
	hipMemcpy(inverse,dev_inverse,size*size*sizeof(float),hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_L);
	hipFree(dev_U);
	hipFree(dev_b);
	hipFree(dev_d);
	hipFree(dev_inverse);

}


/*
int main(int argc, char *argv[]){
	int size;
	
	//checking args and getting args
	if(argc<4){
		perror("Please enter all args eg: ./Inverse mat.txt rows ans.txt");
		return 0;
		}
		
	//char matf1[]=argv[1];
	size=atoi(argv[2]);

	
	//allocating
	float *matA=(float *)malloc(size*size*sizeof(float));
	assert(matA);
	float *ans=(float *)malloc(size*size*sizeof(float));
	assert(ans);
	
	//reading files
	int i,j;
	FILE *fp;
	fp=fopen(argv[1],"r");
	for (i=0;i<size*size;i++){
		fscanf(fp,"%f",&matA[i]);
		}
	fclose(fp);
	printf("reading matrix finished\n");
	//printarray(matA,size,size);
	//printf("\n%d\n",size);
	

	// Inverse
	clock_t start=clock();
	//printarray(tempd,size,size);
	//printarray(matL,size,size);
	//printarray(matb,size,size);
	inverse(ans,matA,size);

	clock_t stop=clock();
    double cputime=(double)((stop-start)/(float)CLOCKS_PER_SEC);
    printf("Time for calculation using CPU is a %1.20f\n",cputime);
	
	//writing to file
	fp=fopen(argv[3],"w");
	for (i=0;i<size;i++){
		for (j=0;j<size;j++){
			fprintf(fp,"%f ",ans[size*i+j]);
		}	
		fprintf(fp,"\n");;
	}
	fclose(fp);
	
	return 0;
}*/
